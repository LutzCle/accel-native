/*
 * Copyright 2018 German Research Center for Artificial Intelligence (DFKI)
 * Author: Clemens Lutz <clemens.lutz@dfki.de>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <hip/hip_runtime.h>
#include <cstdint>

extern "C"
__global__ void add(int32_t a, int32_t b, int64_t* c) {
    *c = a + b;
}

extern "C"
__global__ void vector_add(uint64_t len, int64_t *a, int64_t const* b) {

    for (uint64_t i = 0; i < len; ++i) {
        a[i] = a[i] + b[i];
    }

}
